#include "hip/hip_runtime.h"
#include <stdint.h>
#include <stdlib.h>
#include <string.h>

#include <cutil.h>

#include "collision_check.h"


#define EPSILON 1

__global__ void obstacleKernel(float *cx, float *cy, float *r, float *q_new, uint32_t *d_obs_flag);

__device__ float distance(float cx, float cy, float *q);

__global__ void obstacleKernel(float *cx, float *cy, float *r, float *q_new, uint32_t *d_obs_flag)
{


  const int tid = threadIdx.x;

  __shared__ uint32_t flag;
  flag = 0;


  float d = distance(cx[tid], cy[tid], q_new);
  // printf("%f %f %f\n", cx[tid], cy[tid], r[tid]);

  // if (d < r[tid] + EPSILON)
  // {
  //   printf("collides !!!!!!!!!!!\n");
  //   *d_obs_flag = 1;
  // }


  if (d < r[tid] + EPSILON)
  {
    atomicAdd(&flag, 1);
    // printf("collides !!!!!!!!!!!\n");
    // flag = 1;
  }

  __syncthreads();


  // have one thread write result to global memory
  if (tid == 0)
  {
    if (flag > 0)
    {
      *d_obs_flag = 1;
    }

    else
    {
      *d_obs_flag = 0;
    }
  }
}



__device__ float distance(float cx, float cy, float *q)
{
  float dx = cx - q[0];
  float dy = cy - q[1];

  return sqrt(dx*dx + dy*dy);
}





void obstacle_collision(float *cx, float *cy, float *r, float *q_new, uint32_t *d_obs_flag)
{
  // set flag to 0
  hipMemset(d_obs_flag, 0, sizeof(uint32_t));


  dim3 dimGrid(1);
  dim3 dimBlock(1024);

  obstacleKernel<<<dimGrid, dimBlock>>>(cx, cy, r, q_new, d_obs_flag);

  hipDeviceSynchronize();
}





void copyToDeviceMemory(void* d, void* h, size_t size)
{
	hipMemcpy(d, h, size, hipMemcpyHostToDevice);
}


void copyToHostMemory(void* h, void* d, size_t size)
{
	hipMemcpy(h, d, size, hipMemcpyDeviceToHost);
}


void* allocateDeviceMemory(size_t size)
{
  void *ptr;
  hipMalloc(&ptr, size);
  return ptr;
}


void freeDeviceMemory(void* d)
{
	hipFree(d);
}
